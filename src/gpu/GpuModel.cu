#include "hip/hip_runtime.h"
#include "GpuModel.h"

#define TILE_WIDTH 16

void GPU_Info::printGpuInfo()
{
    hipDeviceProp_t devProv;
	CHECK(hipGetDeviceProperties(&devProv, 0));
	printf("**********GPU info**********\n");
	printf("Name: %s\n", devProv.name);
	printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
	printf("Num SMs: %d\n", devProv.multiProcessorCount);
	printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor);
	printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
	printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
	printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
	printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
	printf("****************************\n");
}

Timer::Timer()
{
	hipEventCreate(&start);
	hipEventCreate(&stop);
}

Timer::~Timer()
{
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

void Timer::Start()
{
	hipEventRecord(start, 0);
	hipEventSynchronize(start);
}

void Timer::Stop()
{
	hipEventRecord(stop, 0);
}

float Timer::Elapsed()
{
	float elapsed;
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	return elapsed;
}

__global__ void kernel_conv_forward_gpu(float* output, const float* input, const float* weight, const int n_sample, const int channel_out, const int channel_in, const int height_in, const int width_in, const int height_kernel)
{
    const int height_out = height_in - height_kernel + 1;
    const int width_out = width_in - height_kernel + 1;
    
    extern __shared__ float shared_data[];
    
    int batch_idx = blockIdx.x;
    int output_feature_idx = blockIdx.y;
    int row_idx = (blockIdx.z / gridDim.z) * blockDim.y + threadIdx.y;
    int col_idx = (blockIdx.z % gridDim.z) * blockDim.x + threadIdx.x;
    
    float accumulator = 0.0f;

    for (int channel_in_idx = 0; channel_in_idx < channel_in; channel_in_idx++)
    {
        for (int kernel_row = 0; kernel_row < height_kernel; kernel_row++)
        {
            for (int kernel_col = 0; kernel_col < height_kernel; kernel_col++)
            {
                int input_row = row_idx + kernel_row;
                int input_col = col_idx + kernel_col;

                // Load input and kernel values into shared memory
                int shared_index = threadIdx.y * blockDim.x + threadIdx.x;
                shared_data[shared_index] = input[(batch_idx * (channel_in * height_in * width_in)) +
                                                (channel_in_idx * (height_in * width_in)) +
                                                (input_row * width_in) +
                                                input_col];

                __syncthreads();

                // Compute convolution with shared memory
                for (int i = 0; i < height_kernel; i++)
                {
                    for (int j = 0; j < height_kernel; j++)
                    {
                        accumulator += shared_data[(threadIdx.y + i) * blockDim.x + (threadIdx.x + j)] *
                                       weight[(output_feature_idx * (channel_in * height_kernel * height_kernel)) +
                                              (channel_in_idx * (height_kernel * height_kernel)) +
                                              (kernel_row * height_kernel) +
                                              kernel_col];
                    }
                }

                __syncthreads();
            }
        }
    }

    if (row_idx < height_out && col_idx < width_out)
    {
        output[(batch_idx * (channel_out * height_out * width_out)) +
               (output_feature_idx * (height_out * width_out)) +
               (row_idx * width_out) +
               col_idx] = accumulator;
    }
}


void GPU_Conv::conv_forward_gpu(float* output, const float* input, const float* weight, const int n_sample, const int channel_out, const int channel_in, const int height_in, const int width_in, const int height_kernel)
{
    const int height_out = height_in - height_kernel + 1;
    const int width_out = width_in - height_kernel + 1;

    // Cấp phát bộ nhớ trên thiết bị
    float *device_input, *device_output, *device_weight;
    hipMalloc((void **)&device_input, n_sample * channel_in * height_in * width_in * sizeof(float));  // Bản đồ đặc trưng đầu vào có kích thước input_channel
    hipMalloc((void **)&device_output, n_sample * channel_out * height_out * width_out * sizeof(float));  // Bản đồ đặc trưng đầu ra có kích thước channel_out
    hipMalloc((void **)&device_weight, channel_out * channel_in * height_kernel * height_kernel * sizeof(float));  // Bộ lọc kích thước input_channel * channel_out có kích thước height_kernel * height_kernel

    // Sao chép dữ liệu đầu vào và trọng số từ máy chủ đến thiết bị
    hipMemcpy(device_input, input, n_sample * channel_in * height_in * width_in * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_weight, weight, channel_out * channel_in * height_kernel * height_kernel * sizeof(float), hipMemcpyHostToDevice);

    // Đặt kích thước grid và block cho kernel và gọi kernel
    int height_grid = ceil(1.0 * height_out / TILE_WIDTH);
    int width_grid = ceil(1.0 * width_out / TILE_WIDTH);
    int Z = height_grid * width_grid;
    dim3 num_threads_per_block(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 num_blocks_in_grid(n_sample, channel_out, Z);

    // Gọi kernel
    kernel_conv_forward_gpu<<<num_blocks_in_grid, num_threads_per_block, TILE_WIDTH * TILE_WIDTH * sizeof(float)>>>(device_output, device_input, device_weight, n_sample, channel_out, channel_in, height_in, width_in, height_kernel);
	CHECK(hipGetLastError());

    // Sao chép kết quả đầu ra từ thiết bị về máy chủ
    hipMemcpy(output, device_output, n_sample * channel_out * height_out * width_out * sizeof(float), hipMemcpyDeviceToHost);

    // Giải phóng bộ nhớ trên thiết bị
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_weight);
}