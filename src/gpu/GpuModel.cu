#include "hip/hip_runtime.h"
#include "GpuModel.h"

#define TILE_WIDTH 28

void GPU_Info::printGpuInfo()
{
    hipDeviceProp_t devProv;
	CHECK(hipGetDeviceProperties(&devProv, 0));
	printf("**********GPU info**********\n");
	printf("Name: %s\n", devProv.name);
	printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
	printf("Num SMs: %d\n", devProv.multiProcessorCount);
	printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor);
	printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
	printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
	printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
	printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
	printf("****************************\n");
}

Timer::Timer()
{
	hipEventCreate(&start);
	hipEventCreate(&stop);
}

Timer::~Timer()
{
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

void Timer::Start()
{
	hipEventRecord(start, 0);
	hipEventSynchronize(start);
}

void Timer::Stop()
{
	hipEventRecord(stop, 0);
}

float Timer::Elapsed()
{
	float elapsed;
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	return elapsed;
}

__global__ void kernel_conv_forward_gpu(float* output, const float* input, const float* weight, const int n_sample, const int channel_out, const int channel_in, const int height_in, const int width_in, const int height_kernel)
{
    // Calculate indices
    const int height_out = height_in - height_kernel + 1;
    const int width_out = width_in - height_kernel + 1;

    int batch_idx = blockIdx.x;
    int output_feature_idx = blockIdx.y;
    int row_idx = (blockIdx.z / gridDim.z) * blockDim.y + threadIdx.y;
    int col_idx = (blockIdx.z % gridDim.z) * blockDim.x + threadIdx.x;

    float accumulator = 0.0f;

    // Loop over input channels, kernel rows, and kernel columns
    for (int channel_in_idx = 0; channel_in_idx < channel_in; channel_in_idx++)
    {
        for (int kernel_row = 0; kernel_row < height_kernel; kernel_row++)
        {
            for (int kernel_col = 0; kernel_col < height_kernel; kernel_col++)
            {
                int input_row = row_idx + kernel_row;
                int input_col = col_idx + kernel_col;

                // Load input values directly from global memory
                int input_index = (batch_idx * (channel_in * height_in * width_in)) + (channel_in_idx * (height_in * width_in)) + (input_row * width_in) + input_col;
                float input_value = input[input_index];

                // Compute convolution with shared memory (weight data)
                accumulator += input_value * weight[(output_feature_idx * (channel_in * height_kernel * height_kernel)) + (channel_in_idx * (height_kernel * height_kernel)) + (kernel_row * height_kernel) + kernel_col];
            }
        }
    }

    // Check bounds before writing to output
    if (row_idx < height_out && col_idx < width_out)
    {
        int output_index = (batch_idx * (channel_out * height_out * width_out)) + (output_feature_idx * (height_out * width_out)) + (row_idx * width_out) + col_idx;

        if (output_index < n_sample * channel_out * height_out * width_out)
        {
            atomicAdd(&output[output_index], accumulator);
        }
    }
}

void GPU_Conv::conv_forward_gpu(float* output, const float* input, const float* weight, const int n_sample, const int channel_out, const int channel_in, const int height_in, const int width_in, const int height_kernel)
{
    // Calculate output size
    const int height_out = height_in - height_kernel + 1;
    const int width_out = width_in - height_kernel + 1;

    // Allocate device memory
    float *device_input, *device_output, *device_weight;
    hipMalloc((void **)&device_input, n_sample * channel_in * height_in * width_in * sizeof(float));
    hipMalloc((void **)&device_output, n_sample * channel_out * height_out * width_out * sizeof(float));
    hipMalloc((void **)&device_weight, channel_out * channel_in * height_kernel * height_kernel * sizeof(float));

    // Copy input and weight data to device
    hipMemcpy(device_input, input, n_sample * channel_in * height_in * width_in * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_weight, weight, channel_out * channel_in * height_kernel * height_kernel * sizeof(float), hipMemcpyHostToDevice);

    // Set grid and block dimensions for kernel and launch it
    dim3 num_threads_per_block(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 num_blocks_in_grid(n_sample, channel_out, ceil(1.0 * height_out / TILE_WIDTH) * ceil(1.0 * width_out / TILE_WIDTH));

    // Launch kernel
    kernel_conv_forward_gpu<<<num_blocks_in_grid, num_threads_per_block>>>(device_output, device_input, device_weight, n_sample, channel_out, channel_in, height_in, width_in, height_kernel);
    CHECK(hipGetLastError());

    // Copy the result back to host
    hipMemcpy(output, device_output, n_sample * channel_out * height_out * width_out * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_weight);
}
