#include "Timer.h"

void GPU_Info::printGpuInfo()
{
    hipDeviceProp_t devProv;
	CHECK(hipGetDeviceProperties(&devProv, 0));
	printf("**********GPU info**********\n");
	printf("Name: %s\n", devProv.name);
	printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
	printf("Num SMs: %d\n", devProv.multiProcessorCount);
	printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor);
	printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
	printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
	printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
	printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
	printf("****************************\n");
}

Timer::Timer()
{
	hipEventCreate(&start);
	hipEventCreate(&stop);
}

Timer::~Timer()
{
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

void Timer::Start()
{
	hipEventRecord(start, 0);
	hipEventSynchronize(start);
}

void Timer::Stop()
{
	hipEventRecord(stop, 0);
}

float Timer::Elapsed()
{
	float elapsed;
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	return elapsed;
}