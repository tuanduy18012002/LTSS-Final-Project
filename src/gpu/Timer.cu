#include "Timer.h"

void GPU_Info::printGpuInfo()
{
    hipDeviceProp_t devProv;
	CHECK(hipGetDeviceProperties(&devProv, 0));
	printf("**********GPU info**********\n");
	printf("Name: %s\n", devProv.name);
	printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
	printf("Num SMs: %d\n", devProv.multiProcessorCount);
	printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor);
	printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
	printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
	printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
	printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
	printf("****************************\n");
}

  GpuTimer::GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	GpuTimer::~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void GpuTimer::Start()
	{
		hipEventRecord(start, 0);
		hipEventSynchronize(start);
	}

	void GpuTimer::Stop()
	{
		hipEventRecord(stop, 0);
	}

	float GpuTimer::Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}